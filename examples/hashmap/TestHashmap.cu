#include "hip/hip_runtime.h"
#include <stdgpu/iterator.h>  // device_begin, device_end
#include <stdgpu/memory.h>    // createDeviceArray, destroyDeviceArray
#include <stdgpu/platform.h>  // STDGPU_HOST_DEVICE
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <iostream>
#include <stdgpu/unordered_map.cuh>  // stdgpu::unordered_map

#include "open3d/core/hashmap/Hashmap.h"
#include "open3d/utility/Console.h"
#include "open3d/utility/Timer.h"

__global__ void insert_numbers(const int* d_keys,
                               const int* d_values,
                               const stdgpu::index_t n,
                               stdgpu::unordered_map<int, int> map) {
    stdgpu::index_t i =
            static_cast<stdgpu::index_t>(blockIdx.x * blockDim.x + threadIdx.x);

    if (i >= n) return;
    map.emplace(d_keys[i], d_values[i]);
}

__global__ void find_numbers(const int* d_keys,
                             int* d_values,
                             const stdgpu::index_t n,
                             stdgpu::unordered_map<int, int> map) {
    stdgpu::index_t i =
            static_cast<stdgpu::index_t>(blockIdx.x * blockDim.x + threadIdx.x);

    if (i >= n) return;
    map.find(d_keys[i])->second;
}

std::pair<std::vector<int>, std::vector<int>> GenerateKVVector(int n,
                                                               int cycle) {
    std::vector<int> k(n), v(n);
    for (int i = 0; i < n; ++i) {
        v[i] = (i % cycle);
        k[i] = v[i] * 100;
    }
    return std::make_pair(k, v);
}

int main(int argc, char** argv) {
    //
    // EXAMPLE DESCRIPTION
    // -------------------
    // This example demonstrates how stdgpu::unordered_map is used to compute a
    // duplicate-free set of numbers.
    //
    using namespace open3d;

    stdgpu::index_t n =
            utility::GetProgramOptionAsInt(argc, argv, "--n", 10000);
    int cycle = utility::GetProgramOptionAsInt(argc, argv, "--cycle", n / 2);
    int runs = utility::GetProgramOptionAsInt(argc, argv, "--runs", 1000);

    auto kv = GenerateKVVector(n, cycle);

    utility::LogInfo("n = {}, cycle = {}", n, cycle);

    // Ours
    core::Tensor t_keys = core::Tensor(kv.first, {n}, core::Dtype::Int32,
                                       core::Device("CUDA:0"));
    core::Tensor t_values = core::Tensor(kv.second, {n}, core::Dtype::Int32,
                                         core::Device("CUDA:0"));

    // Warm up
    core::Device device("CUDA:0");
    {
        core::Hashmap hashmap(n, core::Dtype::Int32, core::Dtype::Int32,
                              core::SizeVector{1}, core::SizeVector{1}, device);
        core::Tensor t_addrs({n}, core::Dtype::Int32, device);
        core::Tensor t_masks({n}, core::Dtype::Bool, device);

        hashmap.Insert(t_keys, t_values, t_addrs, t_masks);
        hashmap.Find(t_keys, t_addrs, t_masks);
        hipDeviceSynchronize();
    }

    utility::Timer timer;

    double insert_time = 0;
    double find_time = 0;
    for (int i = 0; i < runs; ++i) {
        core::Hashmap hashmap(n, core::Dtype::Int32, core::Dtype::Int32,
                              core::SizeVector{1}, core::SizeVector{1}, device);
        core::Tensor t_addrs({n}, core::Dtype::Int32, device);
        core::Tensor t_masks({n}, core::Dtype::Bool, device);

        timer.Start();
        hashmap.Insert(t_keys, t_values, t_addrs, t_masks);
        hipDeviceSynchronize();
        timer.Stop();
        insert_time += timer.GetDuration();

        timer.Start();
        hashmap.Find(t_keys, t_addrs, t_masks);
        hipDeviceSynchronize();
        timer.Stop();
        find_time += timer.GetDuration();

        if (hashmap.Size() != cycle) {
            utility::LogError("ours: incorrect insertion");
        }
    }

    utility::LogInfo("slabhash insertion rate: {}",
                     float(n) / (insert_time / runs));
    utility::LogInfo("slabhash query rate: {}", float(n) / (find_time / runs));

    // stdgpu
    insert_time = 0;
    find_time = 0;
    int* d_keys = static_cast<int*>(t_keys.GetDataPtr());
    int* d_values = static_cast<int*>(t_values.GetDataPtr());
    for (int i = 0; i < runs; ++i) {
        stdgpu::unordered_map<int, int> map =
                stdgpu::unordered_map<int, int>::createDeviceObject(n);
        stdgpu::index_t threads = 128;
        stdgpu::index_t blocks = (n + threads - 1) / threads;

        timer.Start();
        insert_numbers<<<static_cast<unsigned int>(blocks),
                         static_cast<unsigned int>(threads)>>>(d_keys, d_values,
                                                               n, map);
        hipDeviceSynchronize();
        timer.Stop();
        insert_time += timer.GetDuration();

        timer.Start();
        find_numbers<<<static_cast<unsigned int>(blocks),
                       static_cast<unsigned int>(threads)>>>(d_keys, d_values,
                                                             n, map);
        hipDeviceSynchronize();
        timer.Stop();
        find_time += timer.GetDuration();

        if (map.size() != cycle) {
            utility::LogError("stdgpu: incorrect insertion");
        }

        stdgpu::unordered_map<int, int>::destroyDeviceObject(map);
    }
    utility::LogInfo("stdgpu insertion rate: {}",
                     float(n) / (insert_time / runs));
    utility::LogInfo("stdgpu query rate: {}", float(n) / (find_time / runs));
}
